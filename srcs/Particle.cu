#include "hip/hip_runtime.h"
#include "Particle.cuh"


Particle::Particle(int particle_num, float aspect_ratio) {
    this->gravity_pos = glm::vec2(0.0f, 0.0f);
    this->max_distance = sqrt(2);
    initialize_position(particle_num, aspect_ratio);

    // Allocate device memory
    hipMalloc(&cu_position, particle_num * sizeof(glm::vec2));
    hipMalloc(&cu_velocity, particle_num * sizeof(glm::vec2));
    hipMalloc(&cu_color, particle_num * sizeof(glm::vec3));

    hipMemcpy(cu_position, this->position.data(), particle_num * sizeof(glm::vec2), hipMemcpyHostToDevice);
    hipMemcpy(cu_velocity, this->velocity.data(), particle_num * sizeof(glm::vec2), hipMemcpyHostToDevice);
    hipMemcpy(cu_color, this->color.data(), particle_num * sizeof(glm::vec3), hipMemcpyHostToDevice);
}

Particle::~Particle() {
    hipFree(cu_position);
    hipFree(cu_velocity);
    hipFree(cu_color);
}

std::vector<glm::vec2> Particle::get_position() { return this->position; }

std::vector<glm::vec3> Particle::get_color() { return this->color; }

void Particle::set_gravity_pos(float x, float y) {
    this->gravity_pos.x = x;
    this->gravity_pos.y = y;
}

void Particle::initialize_position(int particle_num, float aspect_ratio) {
    std::random_device rd;   // Seed for the random number engine
    std::mt19937 gen(rd());  // Mersenne Twister engine

    // Define a distribution between -1 and 1
    std::uniform_real_distribution<float> dis(0.0f, 2.0f * M_PI);
    std::uniform_real_distribution<float> radius_dis(0.0f, 0.4f);
    for (int i = 0; i < particle_num; i++) {
        glm::vec2 pos;
        float angle = dis(gen);
        float radius = radius_dis(gen);
        pos.x = cos(angle) * aspect_ratio * radius;
        pos.y = sin(angle) * radius;
        this->position.push_back(pos);

        glm::vec3 initial_color(0.0f, 0.0f, 0.0f);
        create_new_color(glm::length(this->gravity_pos - pos), initial_color);
        this->color.push_back(initial_color);
    }

    std::vector<glm::vec2> velo(particle_num, glm::vec2(0.0f, 0.0f));
    this->velocity = velo;
}

// void Particle::update_position_and_color(float delta_time, float aspect_ratio) {
//     glm::vec3 new_color(0.0f, 0.0f, 0.0f);
//     for (std::size_t i = 0; i < this->position.size(); ++i) {
//         glm::vec2 rescaled_pos = this->position[i];
//         rescaled_pos.x /= aspect_ratio;
//         glm::vec2 accel = this->gravity_pos - rescaled_pos;
//         glm::vec2 upscale_accel = accel * glm::length(accel) * 10.0f;

//         this->velocity[i].x += upscale_accel.x * delta_time;
//         this->velocity[i].y += upscale_accel.y * delta_time;
//         this->position[i].x += this->velocity[i].x * delta_time * aspect_ratio;
//         this->position[i].y += this->velocity[i].y * delta_time;

//         create_new_color(glm::length(accel), new_color);
//         this->color[i] = new_color;
//     }
// }

void Particle::update_position_and_color(float delta_time, float aspect_ratio) {
    int threads = 256;
    int blocks = (this->position.size() + threads - 1) / threads;

    update_particle_kernel<<<blocks, threads>>>(
        this->cu_position, this->cu_velocity, this->cu_color, this->gravity_pos,
        delta_time, aspect_ratio, this->position.size(), this->max_distance);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        Particle::~Particle();
        exit(1);
    }
    hipDeviceSynchronize();

    hipMemcpy(this->position.data(), this->cu_position, this->position.size() * sizeof(glm::vec2), hipMemcpyDeviceToHost);
    hipMemcpy(this->color.data(), this->cu_color, this->position.size() * sizeof(glm::vec3), hipMemcpyDeviceToHost);
}

void Particle::create_new_color(float distance, glm::vec3 &new_color) {
    float max_distance = sqrt(2);
    float new_color_val = std::min(distance / max_distance, 1.0f);
    new_color[0] = 1.0f - new_color_val;
    new_color[2] = new_color_val;
}
